#include "hip/hip_runtime.h"
// REQUIRES: nvptx-registered-target

// Make sure we don't allow dynamic initialization for device
// variables, but accept empty constructors allowed by CUDA.

// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device -std=c++11 \
// RUN:     -fno-threadsafe-statics -emit-llvm -o - %s | FileCheck --check-prefixes=CHECK,NVPTX %s

// RUN: %clang_cc1 -triple amdgcn -fcuda-is-device -std=c++11 \
// RUN:     -fno-threadsafe-statics -emit-llvm -o - %s | FileCheck --check-prefixes=CHECK,AMDGCN %s

#ifdef __clang__
#include "Inputs/hip/hip_runtime.h"
#endif

// Use the types we share with Sema tests.
#include "Inputs/cuda-initializers.h"

__device__ int d_v;
// CHECK: @d_v = addrspace(1) externally_initialized global i32 0,
__shared__ int s_v;
// CHECK: @s_v = addrspace(3) global i32 undef,
__constant__ int c_v;
// NVPTX: addrspace(4) externally_initialized global i32 0,
// AMDGCN: addrspace(2) externally_initialized global i32 0,

__device__ int d_v_i = 1;
// CHECK: @d_v_i = addrspace(1) externally_initialized global i32 1,

// trivial constructor -- allowed
__device__ T d_t;
// CHECK: @d_t = addrspace(1) externally_initialized global %struct.T zeroinitializer
__shared__ T s_t;
// CHECK: @s_t = addrspace(3) global %struct.T undef,
__constant__ T c_t;
// NVPTX: @c_t = addrspace(4) externally_initialized global %struct.T zeroinitializer,
// AMDGCN: @c_t = addrspace(2) externally_initialized global %struct.T zeroinitializer,

__device__ T d_t_i = {2};
// CHECK: @d_t_i = addrspace(1) externally_initialized global %struct.T { i32 2 },
__constant__ T c_t_i = {2};
// NVPTX: @c_t_i = addrspace(4) externally_initialized global %struct.T { i32 2 },
// AMDGCN: @c_t_i = addrspace(2) externally_initialized global %struct.T { i32 2 },

// empty constructor
__device__ EC d_ec;
// CHECK: @d_ec = addrspace(1) externally_initialized global %struct.EC zeroinitializer,
__shared__ EC s_ec;
// CHECK: @s_ec = addrspace(3) global %struct.EC undef,
__constant__ EC c_ec;
// NVPTX: @c_ec = addrspace(4) externally_initialized global %struct.EC zeroinitializer,
// AMDGCN: @c_ec = addrspace(2) externally_initialized global %struct.EC zeroinitializer,

// empty destructor
__device__ ED d_ed;
// CHECK: @d_ed = addrspace(1) externally_initialized global %struct.ED zeroinitializer,
__shared__ ED s_ed;
// CHECK: @s_ed = addrspace(3) global %struct.ED undef,
__constant__ ED c_ed;
// NVPTX: @c_ed = addrspace(4) externally_initialized global %struct.ED zeroinitializer,
// AMDGCN: @c_ed = addrspace(2) externally_initialized global %struct.ED zeroinitializer,

__device__ ECD d_ecd;
// CHECK: @d_ecd = addrspace(1) externally_initialized global %struct.ECD zeroinitializer,
__shared__ ECD s_ecd;
// CHECK: @s_ecd = addrspace(3) global %struct.ECD undef,
__constant__ ECD c_ecd;
// NVPTX: @c_ecd = addrspace(4) externally_initialized global %struct.ECD zeroinitializer,
// AMDGCN: @c_ecd = addrspace(2) externally_initialized global %struct.ECD zeroinitializer,

// empty templated constructor -- allowed with no arguments
__device__ ETC d_etc;
// CHECK: @d_etc = addrspace(1) externally_initialized global %struct.ETC zeroinitializer,
__shared__ ETC s_etc;
// CHECK: @s_etc = addrspace(3) global %struct.ETC undef,
__constant__ ETC c_etc;
// NVPTX: @c_etc = addrspace(4) externally_initialized global %struct.ETC zeroinitializer,
// AMDGCN: @c_etc = addrspace(2) externally_initialized global %struct.ETC zeroinitializer,

__device__ NCFS d_ncfs;
// CHECK: @d_ncfs = addrspace(1) externally_initialized global %struct.NCFS { i32 3 }
__constant__ NCFS c_ncfs;
// NVPTX: @c_ncfs = addrspace(4) externally_initialized global %struct.NCFS { i32 3 }
// AMDGCN: @c_ncfs = addrspace(2) externally_initialized global %struct.NCFS { i32 3 }

// Regular base class -- allowed
__device__ T_B_T d_t_b_t;
// CHECK: @d_t_b_t = addrspace(1) externally_initialized global %struct.T_B_T zeroinitializer,
__shared__ T_B_T s_t_b_t;
// CHECK: @s_t_b_t = addrspace(3) global %struct.T_B_T undef,
__constant__ T_B_T c_t_b_t;
// NVPTX: @c_t_b_t = addrspace(4) externally_initialized global %struct.T_B_T zeroinitializer,
// AMDGCN: @c_t_b_t = addrspace(2) externally_initialized global %struct.T_B_T zeroinitializer,

// Incapsulated object of allowed class -- allowed
__device__ T_F_T d_t_f_t;
// CHECK: @d_t_f_t = addrspace(1) externally_initialized global %struct.T_F_T zeroinitializer,
__shared__ T_F_T s_t_f_t;
// CHECK: @s_t_f_t = addrspace(3) global %struct.T_F_T undef,
__constant__ T_F_T c_t_f_t;
// NVPTX: @c_t_f_t = addrspace(4) externally_initialized global %struct.T_F_T zeroinitializer,
// AMDGCN: @c_t_f_t = addrspace(2) externally_initialized global %struct.T_F_T zeroinitializer,

// array of allowed objects -- allowed
__device__ T_FA_T d_t_fa_t;
// CHECK: @d_t_fa_t = addrspace(1) externally_initialized global %struct.T_FA_T zeroinitializer,
__shared__ T_FA_T s_t_fa_t;
// CHECK: @s_t_fa_t = addrspace(3) global %struct.T_FA_T undef,
__constant__ T_FA_T c_t_fa_t;
// NVPTX: @c_t_fa_t = addrspace(4) externally_initialized global %struct.T_FA_T zeroinitializer,
// AMDGCN: @c_t_fa_t = addrspace(2) externally_initialized global %struct.T_FA_T zeroinitializer,


// Calling empty base class initializer is OK
__device__ EC_I_EC d_ec_i_ec;
// CHECK: @d_ec_i_ec = addrspace(1) externally_initialized global %struct.EC_I_EC zeroinitializer,
__shared__ EC_I_EC s_ec_i_ec;
// CHECK: @s_ec_i_ec = addrspace(3) global %struct.EC_I_EC undef,
__constant__ EC_I_EC c_ec_i_ec;
// NVPTX: @c_ec_i_ec = addrspace(4) externally_initialized global %struct.EC_I_EC zeroinitializer,
// AMDGCN: @c_ec_i_ec = addrspace(2) externally_initialized global %struct.EC_I_EC zeroinitializer,

// We should not emit global initializers for device-side variables.
// CHECK-NOT: @__cxx_global_var_init

// Make sure that initialization restrictions do not apply to local
// variables.
__device__ void df() {
  T t;
  // CHECK-NOT: call
  EC ec;
  // NVPTX:   call void @_ZN2ECC1Ev(%struct.EC* %ec)
  // AMDGCN:   call void @_ZN2ECC1Ev(%struct.EC addrspace(4)* %ec)
  ED ed;
  // CHECK-NOT: call
  ECD ecd;
  // NVPTX:   call void @_ZN3ECDC1Ev(%struct.ECD* %ecd)
  // AMDGCN:   call void @_ZN3ECDC1Ev(%struct.ECD addrspace(4)* %ecd)
  ETC etc;
  // NVPTX:   call void @_ZN3ETCC1IJEEEDpT_(%struct.ETC* %etc)
  // AMDGCN:   call void @_ZN3ETCC1IJEEEDpT_(%struct.ETC addrspace(4)* %etc)
  UC uc;
  // undefined constructor -- not allowed
  // NVPTX:   call void @_ZN2UCC1Ev(%struct.UC* %uc)
  // AMDGCN:   call void @_ZN2UCC1Ev(%struct.UC addrspace(4)* %uc)
  UD ud;
  // undefined destructor -- not allowed
  // CHECK-NOT: call
  ECI eci;
  // empty constructor w/ initializer list -- not allowed
  // NVPTX:   call void @_ZN3ECIC1Ev(%struct.ECI* %eci)
  // AMDGCN:   call void @_ZN3ECIC1Ev(%struct.ECI addrspace(4)* %eci)
  NEC nec;
  // non-empty constructor -- not allowed
  // NVPTX:   call void @_ZN3NECC1Ev(%struct.NEC* %nec)
  // AMDGCN:   call void @_ZN3NECC1Ev(%struct.NEC addrspace(4)* %nec)
  // non-empty destructor -- not allowed
  NED ned;
  // no-constructor,  virtual method -- not allowed
  // NVPTX:   call void @_ZN3NCVC1Ev(%struct.NCV* %ncv)
  // AMDGCN:   call void @_ZN3NCVC1Ev(%struct.NCV addrspace(4)* %ncv)
  NCV ncv;
  // CHECK-NOT: call
  VD vd;
  // NVPTX:   call void @_ZN2VDC1Ev(%struct.VD* %vd)
  // AMDGCN:   call void @_ZN2VDC1Ev(%struct.VD addrspace(4)* %vd)
  NCF ncf;
  // NVPTX:   call void @_ZN3NCFC1Ev(%struct.NCF* %ncf)
  // AMDGCN:   call void @_ZN3NCFC1Ev(%struct.NCF addrspace(4)* %ncf)
  NCFS ncfs;
  // NVPTX:   call void @_ZN4NCFSC1Ev(%struct.NCFS* %ncfs)
  // AMDGCN:   call void @_ZN4NCFSC1Ev(%struct.NCFS addrspace(4)* %ncfs)
  UTC utc;
  // NVPTX:   call void @_ZN3UTCC1IJEEEDpT_(%struct.UTC* %utc)
  // AMDGCN:   call void @_ZN3UTCC1IJEEEDpT_(%struct.UTC addrspace(4)* %utc)
  NETC netc;
  // NVPTX:   call void @_ZN4NETCC1IJEEEDpT_(%struct.NETC* %netc)
  // AMDGCN:   call void @_ZN4NETCC1IJEEEDpT_(%struct.NETC addrspace(4)* %netc)
  T_B_T t_b_t;
  // CHECK-NOT: call
  T_F_T t_f_t;
  // CHECK-NOT: call
  T_FA_T t_fa_t;
  // CHECK-NOT: call
  EC_I_EC ec_i_ec;
  // NVPTX:   call void @_ZN7EC_I_ECC1Ev(%struct.EC_I_EC* %ec_i_ec)
  // AMDGCN:   call void @_ZN7EC_I_ECC1Ev(%struct.EC_I_EC addrspace(4)* %ec_i_ec)
  EC_I_EC1 ec_i_ec1;
  // NVPTX:   call void @_ZN8EC_I_EC1C1Ev(%struct.EC_I_EC1* %ec_i_ec1)
  // AMDGCN:   call void @_ZN8EC_I_EC1C1Ev(%struct.EC_I_EC1 addrspace(4)* %ec_i_ec1)
  T_V_T t_v_t;
  // NVPTX:   call void @_ZN5T_V_TC1Ev(%struct.T_V_T* %t_v_t)
  // AMDGCN:   call void @_ZN5T_V_TC1Ev(%struct.T_V_T addrspace(4)* %t_v_t)
  T_B_NEC t_b_nec;
  // NVPTX:   call void @_ZN7T_B_NECC1Ev(%struct.T_B_NEC* %t_b_nec)
  // AMDGCN:   call void @_ZN7T_B_NECC1Ev(%struct.T_B_NEC addrspace(4)* %t_b_nec)
  T_F_NEC t_f_nec;
  // NVPTX:   call void @_ZN7T_F_NECC1Ev(%struct.T_F_NEC* %t_f_nec)
  // AMDGCN:   call void @_ZN7T_F_NECC1Ev(%struct.T_F_NEC addrspace(4)* %t_f_nec)
  T_FA_NEC t_fa_nec;
  // NVPTX:   call void @_ZN8T_FA_NECC1Ev(%struct.T_FA_NEC* %t_fa_nec)
  // AMDGCN:   call void @_ZN8T_FA_NECC1Ev(%struct.T_FA_NEC addrspace(4)* %t_fa_nec)
  T_B_NED t_b_ned;
  // CHECK-NOT: call
  T_F_NED t_f_ned;
  // CHECK-NOT: call
  T_FA_NED t_fa_ned;
  // CHECK-NOT: call
  static __shared__ EC s_ec;
  // NVPTX-NOT: call void @_ZN2ECC1Ev(%struct.EC* addrspacecast (%struct.EC addrspace(3)* @_ZZ2dfvE4s_ec to %struct.EC*))
  // AMDGCN-NOT: call void @_ZN2ECC1Ev(%struct.EC addrspace(4)* addrspacecast (%struct.EC addrspace(3)* @_ZZ2dfvE4s_ec to %struct.EC addrspace(4)*))
  static __shared__ ETC s_etc;
  // NVPTX-NOT: call void @_ZN3ETCC1IJEEEDpT_(%struct.ETC* addrspacecast (%struct.ETC addrspace(3)* @_ZZ2dfvE5s_etc to %struct.ETC*))
  // AMDGCN-NOT: call void @_ZN3ETCC1IJEEEDpT_(%struct.ETC addrspace(4)* addrspacecast (%struct.ETC addrspace(3)* @_ZZ2dfvE5s_etc to %struct.ETC addrspace(4)*))

  // anchor point separating constructors and destructors
  df(); // CHECK: call void @_Z2dfv()

  // Verify that we only call non-empty destructors
  // NVPTX-NEXT: call void @_ZN8T_FA_NEDD1Ev(%struct.T_FA_NED* %t_fa_ned)
  // NVPTX-NEXT: call void @_ZN7T_F_NEDD1Ev(%struct.T_F_NED* %t_f_ned)
  // NVPTX-NEXT: call void @_ZN7T_B_NEDD1Ev(%struct.T_B_NED* %t_b_ned)
  // NVPTX-NEXT: call void @_ZN2VDD1Ev(%struct.VD* %vd)
  // NVPTX-NEXT: call void @_ZN3NEDD1Ev(%struct.NED* %ned)
  // NVPTX-NEXT: call void @_ZN2UDD1Ev(%struct.UD* %ud)
  // NVPTX-NEXT: call void @_ZN3ECDD1Ev(%struct.ECD* %ecd)
  // NVPTX-NEXT: call void @_ZN2EDD1Ev(%struct.ED* %ed)

  // AMDGCN-NEXT: call void @_ZN8T_FA_NEDD1Ev(%struct.T_FA_NED addrspace(4)* %t_fa_ned)
  // AMDGCN-NEXT: call void @_ZN7T_F_NEDD1Ev(%struct.T_F_NED addrspace(4)* %t_f_ned)
  // AMDGCN-NEXT: call void @_ZN7T_B_NEDD1Ev(%struct.T_B_NED addrspace(4)* %t_b_ned)
  // AMDGCN-NEXT: call void @_ZN2VDD1Ev(%struct.VD addrspace(4)* %vd)
  // AMDGCN-NEXT: call void @_ZN3NEDD1Ev(%struct.NED addrspace(4)* %ned)
  // AMDGCN-NEXT: call void @_ZN2UDD1Ev(%struct.UD addrspace(4)* %ud)
  // AMDGCN-NEXT: call void @_ZN3ECDD1Ev(%struct.ECD addrspace(4)* %ecd)
  // AMDGCN-NEXT: call void @_ZN2EDD1Ev(%struct.ED addrspace(4)* %ed)

  // CHECK-NEXT: ret void
}

// We should not emit global init function.
// CHECK-NOT: @_GLOBAL__sub_I
