#include "hip/hip_runtime.h"
// RUN: %clang_cc1 --std=c++11 -triple x86_64-unknown-linux-gnu -emit-llvm -o - %s | \
// RUN:  FileCheck -check-prefixes=HOST,CHECK %s

// RUN: %clang_cc1 --std=c++11 -fcuda-is-device -triple nvptx64-nvidia-cuda \
// RUN:   -emit-llvm -o - %s | FileCheck -check-prefixes=DEVICE,CHECK,NVPTX %s

// RUN: %clang_cc1 --std=c++11 -fcuda-is-device -triple amdgcn-amd-amdhsa \
// RUN:   -emit-llvm -o - %s -DAMDGCN| FileCheck -check-prefixes=DEVICE,CHECK,AMDGCN %s

#include "Inputs/hip/hip_runtime.h"

struct U {
  short x;
} __attribute__((packed));

struct S {
  int *ptr;
  char a;
  U u;
};

// Clang should generate a packed LLVM struct for S (denoted by the <>s),
// otherwise this test isn't interesting.
// HOST: %struct.S = type <{ i32*, i8, %struct.U, [5 x i8] }>
// NVPTX: %struct.S = type <{ i32*, i8, %struct.U, [5 x i8] }>
// AMDGCN: %struct.S = type <{ i32 addrspace(4)*, i8, %struct.U, [5 x i8] }>

static_assert(alignof(S) == 8, "Unexpected alignment.");

// HOST-LABEL: @_Z6kernelc1SPi
// Marshalled kernel args should be:
//   1. offset 0, width 1
//   2. offset 8 (because alignof(S) == 8), width 16
//   3. offset 24, width 8
// HOST: call i32 @hipSetupArgument({{[^,]*}}, i64 1, i64 0)
// HOST: call i32 @hipSetupArgument({{[^,]*}}, i64 16, i64 8)
// HOST: call i32 @hipSetupArgument({{[^,]*}}, i64 8, i64 24)

// DEVICE-LABEL: @_Z6kernelc1SPi
// NVPTX-SAME: i8{{[^,]*}}, %struct.S* byval align 8{{[^,]*}}, i32*
// AMDGCN-SAME: i8{{[^,]*}}, %struct.S* byval align 8{{[^,]*}}, i32 addrspace(4)*
__global__ void kernel(char a, S s, int *b) {}
