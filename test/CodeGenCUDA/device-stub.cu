#include "hip/hip_runtime.h"
// RUN: echo "GPU binary would be here" > %t
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -fcuda-include-gpubinary %t -o - | FileCheck -check-prefixes=CHECK,CUDA %s
// RUN: %clang_cc1 -triple x86_64-linux-gnu -emit-llvm %s -fcuda-include-gpubinary %t -o -  -DNOGLOBALS \
// RUN:   | FileCheck %s -check-prefix=NOGLOBALS
// RUN: %clang_cc1 -triple x86_64-linux-gnu -x hip -emit-llvm %s -o - | FileCheck %s -check-prefix=NOGPUBIN
// RUN: %clang_cc1 -triple x86_64-linux-gnu -x hip -emit-llvm %s -fcuda-include-gpubinary %t -o - | FileCheck -check-prefixes=CHECK,HIP %s
// RUN: %clang_cc1 -triple x86_64-linux-gnu -x hip -emit-llvm %s -fcuda-include-gpubinary %t -o -  -DNOGLOBALS \
// RUN:   | FileCheck %s -check-prefix=NOGLOBALS
// RUN: %clang_cc1 -triple x86_64-linux-gnu -x hip -emit-llvm %s -o - | FileCheck %s -check-prefix=NOGPUBIN

#include "Inputs/hip/hip_runtime.h"

#ifndef NOGLOBALS
// CHECK-DAG: @device_var = internal global i32
__device__ int device_var;

// CHECK-DAG: @constant_var = internal global i32
__constant__ int constant_var;

// CHECK-DAG: @shared_var = internal global i32
__shared__ int shared_var;

// Make sure host globals don't get internalized...
// CHECK-DAG: @host_var = global i32
int host_var;
// ... and that extern vars remain external.
// CHECK-DAG: @ext_host_var = external global i32
extern int ext_host_var;

// Shadows for external device-side variables are *definitions* of
// those variables.
// CHECK-DAG: @ext_device_var = internal global i32
extern __device__ int ext_device_var;
// CHECK-DAG: @ext_device_var = internal global i32
extern __constant__ int ext_constant_var;

void use_pointers() {
  int *p;
  p = &device_var;
  p = &constant_var;
  p = &shared_var;
  p = &host_var;
  p = &ext_device_var;
  p = &ext_constant_var;
  p = &ext_host_var;
}

// Make sure that all parts of GPU code init/cleanup are there:
// * constant unnamed string with the kernel name
// CHECK: private unnamed_addr constant{{.*}}kernelfunc{{.*}}\00"
// * constant unnamed string with GPU binary
// CHECK: private unnamed_addr constant{{.*GPU binary would be here.*}}\00"
// CHECK-SAME: section ".nv_fatbin", align 8
// * constant struct that wraps GPU binary
// CUDA: @__[[PREFIX:cuda]]_fatbin_wrapper = internal constant { i32, i32, i8*, i8* }
// HIP: @__[[PREFIX:hip]]_fatbin_wrapper = internal constant { i32, i32, i8*, i8* }
// CHECK-SAME: { i32 1180844977, i32 1, {{.*}}, i8* null }
// CHECK-SAME: section ".nvFatBinSegment"
// * variable to save GPU binary handle after initialization
// CHECK: @__[[PREFIX]]_gpubin_handle = internal global i8** null
// * Make sure our constructor/destructor was added to global ctor/dtor list.
// CHECK: @llvm.global_ctors = appending global {{.*}}@__[[PREFIX]]_module_ctor
// CHECK: @llvm.global_dtors = appending global {{.*}}@__[[PREFIX]]_module_dtor

// Test that we build the correct number of calls to hipSetupArgument followed
// by a call to hipLaunchByPtr.

// CHECK: define{{.*}}kernelfunc
// CHECK: call{{.*}}[[PREFIX]]SetupArgument
// CHECK: call{{.*}}[[PREFIX]]SetupArgument
// CHECK: call{{.*}}[[PREFIX]]SetupArgument
// CHECK: call{{.*}}[[PREFIX]]Launch
__global__ void kernelfunc(int i, int j, int k) {}

// Test that we've built correct kernel launch sequence.
// CHECK: define{{.*}}hostfunc
// CHECK: call{{.*}}[[PREFIX]]ConfigureCall
// CHECK: call{{.*}}kernelfunc
void hostfunc(void) { kernelfunc<<<1, 1>>>(1, 1, 1); }
#endif

// Test that we've built a function to register kernels and global vars.
// CHECK: define internal void @__[[PREFIX]]_register_globals
// CHECK: call{{.*}}[[PREFIX]]RegisterFunction(i8** %0, {{.*}}kernelfunc
// CHECK-DAG: call{{.*}}[[PREFIX]]RegisterVar(i8** %0, {{.*}}device_var{{.*}}i32 0, i32 4, i32 0, i32 0
// CHECK-DAG: call{{.*}}[[PREFIX]]RegisterVar(i8** %0, {{.*}}constant_var{{.*}}i32 0, i32 4, i32 1, i32 0
// CHECK-DAG: call{{.*}}[[PREFIX]]RegisterVar(i8** %0, {{.*}}ext_device_var{{.*}}i32 1, i32 4, i32 0, i32 0
// CHECK-DAG: call{{.*}}[[PREFIX]]RegisterVar(i8** %0, {{.*}}ext_constant_var{{.*}}i32 1, i32 4, i32 1, i32 0
// CHECK: ret void

// Test that we've built contructor..
// CHECK: define internal void @__[[PREFIX]]_module_ctor
//   .. that calls __[[PREFIX]]RegisterFatBinary(&__[[PREFIX]]_fatbin_wrapper)
// CHECK: call{{.*}}[[PREFIX]]RegisterFatBinary{{.*}}__[[PREFIX]]_fatbin_wrapper
//   .. stores return value in __[[PREFIX]]_gpubin_handle
// CHECK-NEXT: store{{.*}}__[[PREFIX]]_gpubin_handle
//   .. and then calls __[[PREFIX]]_register_globals
// CHECK-NEXT: call void @__[[PREFIX]]_register_globals

// Test that we've created destructor.
// CHECK: define internal void @__[[PREFIX]]_module_dtor
// CHECK: load{{.*}}__[[PREFIX]]_gpubin_handle
// CHECK-NEXT: call void @__[[PREFIX]]UnregisterFatBinary

// There should be no __[[PREFIX]]_register_globals if we have no
// device-side globals, but we still need to register GPU binary.
// Skip GPU binary string first.
// NOGLOBALS: @0 = private unnamed_addr constant{{.*}}
// NOGLOBALS-NOT: define internal void @__{{.*}}_register_globals
// NOGLOBALS: define internal void @__[[PREFIX:.*]]_module_ctor
// NOGLOBALS: call{{.*}}[[PREFIX]]RegisterFatBinary{{.*}}__[[PREFIX]]_fatbin_wrapper
// NOGLOBALS-NOT: call void @__[[PREFIX]]_register_globals
// NOGLOBALS: define internal void @__[[PREFIX]]_module_dtor
// NOGLOBALS: call void @__[[PREFIX]]UnregisterFatBinary

// There should be no constructors/destructors if we have no GPU binary.
// NOGPUBIN-NOT: define internal void @__[[PREFIX]]_register_globals
// NOGPUBIN-NOT: define internal void @__[[PREFIX]]_module_ctor
// NOGPUBIN-NOT: define internal void @__[[PREFIX]]_module_dtor
