// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -triple amdgcn -fcuda-is-device -std=c++11 -x hip \
// RUN:     -fno-threadsafe-statics -emit-llvm -o - %s | FileCheck -check-prefixes=DEVICE,AMDGCN %s

//#include "Inputs/hip/hip_runtime.h"

#define __device__ __attribute__((device))
__device__ void f();
struct A {
  __device__ A() { f(); }
};

__device__ A a;
