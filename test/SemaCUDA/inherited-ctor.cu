
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -std=c++11 -fsyntax-only -verify %s
// expected-no-diagnostics

struct A {
  A(const int &x) {}
};

struct B : A {
  using A::A;
};

struct C {
  struct B b;
  C() : b(0) {}
};

void test() {
  B b(0);
  C c;
}
